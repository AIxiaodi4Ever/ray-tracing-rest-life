#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#define STB_IMAGE_IMPLEMENTATION        // ����STB_IMAGE_IMPLEMENTATION��ͷ�ļ�ֻ������������Դ�룬���ڽ�ͷ�ļ���Ϊ.cpp�ļ�
                                        // �������"�޷��������ⲿ���� stbi_load������ main �������˸÷���"
#include "./stb-master/stb_image.h"
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "aarect.h"
#include "box.h"
#include "moving_sphere.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"
#include "pdf.h"

using namespace std;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, const char *const func, const char *const file, const int line)
{
    if (result)
    {
        cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << "'" << func << "\n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 ray_color(const ray& r, const vec3& background, hittable **d_world, hittable **shape_list, hiprandState *local_rand_state)
{
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    vec3 cur_emitted = vec3(0, 0, 0);
    scatter_record srec;
    for(int i = 0; i < 50; i++) 
    {
        hit_record rec;
        if ((*d_world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) 
        {
            float pdf_val;
            vec3 emitted = rec.mat_ptr->emitted(cur_ray, rec, rec.u, rec.v, rec.p);
            if(rec.mat_ptr->scatter(cur_ray, rec, srec, local_rand_state)) 
            {
                cur_emitted = cur_emitted + cur_attenuation * emitted;
                if (srec.is_specular)
                {
                    cur_attenuation =  cur_attenuation * srec.attenuation;
                    cur_ray = srec.specular_ray;
                }
                else
                {
                    // �޸�pdf��scattered
                    //hittable_pdf p0(*light_shape, rec.p);
                    hittable_pdf p0(*shape_list, rec.p);
                    mixture_pdf p(&p0, srec.pdf_ptr);
                    // ����ȷ�����䷽���Ƿ������ڲ���ͨ��rec.mat_ptr->scattering_pdf���м��ģ�����������ڲ����򷵻�0
                    ray scattered = ray(rec.p, p.generate(local_rand_state), cur_ray.time());
                    pdf_val = p.value(scattered.direction());
                    // this delete can't ignore, otherwise you will encount CUDA error = 700
                    delete srec.pdf_ptr;
                    cur_attenuation = cur_attenuation * srec.attenuation * rec.mat_ptr->scattering_pdf(cur_ray, rec, scattered) / pdf_val;
                    cur_ray = scattered;
                }
            }
            else    
            {
                if (i == 0)
                    return emitted;
                return (cur_emitted + cur_attenuation * emitted); 
            }
        }
        else 
        {
            if (i == 0)
                return background;
            return cur_emitted + cur_attenuation * background;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) 
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(2020, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i > max_x || j > max_y) return;
    int pixel_index = j * max_x + i;
    // Each thread get same seed, a different sequence number, no offset
    hiprand_init(2020, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam,
                hittable **d_world, hittable **shape_list, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = (j * max_x + i);
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    vec3 background(0, 0, 0);
    for (int s = 0; s < ns; ++s)
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        vec3 temp = ray_color(r, background, d_world, shape_list, &local_rand_state);
        /* ����NaN */
        // ���������������������ۼӺ��ٸ�ֵ0.0ֻ�ܵõ���ɫ�����
        // ������ֻ�ܽ��NaN���µĺ�ɫ��㣬������˰�ɫ���
        if (!(temp[0] == temp[0])) temp[0] = 0.0;
        if (!(temp[1] == temp[1])) temp[1] = 0.0;
        if (!(temp[2] == temp[2])) temp[2] = 0.0;
        // ������������µİ�ɫ��㣬100�������ģ�Ӧ��ֻҪ���ڹ�Դ��ֵ����
        if (temp[0] > 100)  temp[0] = 0.0;
        if (temp[1] > 100)  temp[1] = 0.0;
        if (temp[2] > 100)  temp[2] = 0.0;
        col += temp;
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);      // gamma corrected
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
    /*printf("block index : %d, %d\n", blockIdx.x, blockIdx.y);
    printf("thread index : %d, %d\n", threadIdx.x, threadIdx.y);*/
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera, 
            int nx, int ny, hiprandState *rand_state, unsigned char* d_texture_data, int* d_inxyn) 
{
    // d_world��һ��hittable_list���̳���hittable���洢�����еĿ���ײ���壬ͨ������d_world��hit����ʵ�ֶ���������ı������ҵ�������壬
    // �ṩ��һ����󣬷������Ҫ��ray_color���ֶ�չ�����б���
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {
        hiprandState local_rand_state = *rand_state;

        material* red = new lambertian(new const_texture(vec3(0.65, 0.05, 0.05)));
        material* white = new lambertian(new const_texture(vec3(0.73, 0.73, 0.73)));
        material* green = new lambertian(new const_texture(vec3(0.12, 0.45, 0.15)));

        material* light = new diffuse_light(new const_texture(vec3(15, 15, 15)));   // (15,15,15)

        material* ima = new lambertian(new image_texture(d_texture_data, d_inxyn[0], d_inxyn[1]));
        material* noise = new lambertian(new noise_texture(&local_rand_state, 0.2));

        material* aluminum = new metal(vec3(0.8, 0.85, 0.88), 0.0);

        material* glass = new dielectric(3);

        d_list[0] = new flip_face(new yz_rect(aluminum, 0, 555, 0, 555, 555)); // green
        d_list[1] = new yz_rect(aluminum, 0, 555, 0, 555, 0);    // red
        // flip_face�������Ǳ�֤��Դ����
        d_list[2] = new flip_face(new xz_rect(light, 213, 343, 227, 332, 554));    // ��Դ (150, 400, 150, 400, 554)
        d_list[3] = new xz_rect(white, 0, 555, 0, 555, 0);  // white
        d_list[4] = new flip_face(new xz_rect(white, 0, 555, 0, 555, 555)); // white
        d_list[5] = new flip_face(new xy_rect(ima, 0, 555, 0, 555, 555)); // white

        // ����ת��ƽ�ƣ������޷��õ���ȷ��λ�ã�ԭ����ת����������y��������Ҫ������Ϊ��ת��������������غϣ�
        /*hittable* box1 = new box(white, vec3(0, 0, 0), vec3(165, 165, 165));    /// (0,0,0) (165,165,165)
        box1 = new rotate_y(box1, -18);
        d_list[6] = new translate(box1, vec3(130, 0, 65));  //(130,0,65)*/
        //hittable *sphere = new sphere(aluminum, vec3(190, 90, 190), 90);
        d_list[6] = new sphere(glass, vec3(190, 90, 190), 90);
        hittable* box2 = new box(glass, vec3(0, 0, 0), vec3(165, 330, 165));      /// (0,0,0) (165,330,165)
        box2 = new rotate_y(box2, 15);
        d_list[7] = new translate(box2, vec3(265, 0.1, 295)); // (265,0,295)

        *rand_state = local_rand_state;
        *d_world  = new hittable_list(d_list, 8);

        vec3 lookfrom(278, 278, -800);  // 278, 278, -800
        vec3 lookat(278 , 278, 0);
        float dist_to_focus = 1355; (lookfrom-lookat).length();
        float aperture = 5.0; //0.1
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 40,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus,
                                 0.0,
                                 1.0);
    }
}

__global__ void create_shape(hittable **shape, hittable **shape_list, int num_shape)
{
    shape[0] = new xz_rect(0, 213, 343, 227, 332, 554);
    shape[1] = new sphere(0, vec3(190, 90, 190), 90);
    *shape_list = new hittable_list(shape, 2);
}

__global__ void free_world(hittable **d_list, hittable **d_world, camera **d_camera, hittable **shape)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {
        /*for (int i = 0; i < 8; ++i)
        {
            delete d_list[i]->mat_ptr;
            delete d_list[i];
        }*/
        delete *d_world;
        delete *d_camera;
        delete *shape;
    }
}

int main()
{
    const int nx = 1200;
    const int ny = 1200;
    const int ns = 777;     // ÿ��������������(�����)
    int tx = 16, ty = 16;

    cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    cerr << "in " << tx << "x" << ty << " threads.\n";

    const int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged(&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMallocManaged(&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMallocManaged(&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1, 1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // ֻ����host���������stbi_load����ȡͼƬ
    int inx, iny, inn;
    int *d_inxyn;
    // string����ת����const char*
    //string image_name = "IMG_20200910_000256.jpg"
    //"60847663_p0.jpg"
    unsigned char* texture_data = stbi_load("60847663_p0.jpg", &inx, &iny, &inn, 0);
    unsigned char* d_texture_data;
    // ����ͼ��
    checkCudaErrors(hipMallocManaged(&d_texture_data, inx * iny * inn * sizeof(unsigned char)));
    checkCudaErrors(hipMemcpy(d_texture_data, texture_data, inx * iny * inn * sizeof(unsigned char), hipMemcpyHostToDevice));
    // ����ͼ��ߴ缰ͨ����
    checkCudaErrors(hipMallocManaged(&d_inxyn, 3 * sizeof(int)));
    checkCudaErrors(hipMemcpy(d_inxyn, &inx, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_inxyn + 1, &iny, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_inxyn + 2, &inn, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hittables & camera
    hittable **d_list;      
    hittable **d_world;     // ʹ��ָ���ָ���ԭ���ǣ��������ָ���creat_world��ᴫ�ݸ��������¸���ʱ��������Ч�ռ�
    camera **d_camera;
    int num_hittables = 8;
    checkCudaErrors(hipMallocManaged(&d_list, num_hittables * sizeof(hittable *)));
    checkCudaErrors(hipMallocManaged(&d_world, sizeof(hittable *)));
    checkCudaErrors(hipMallocManaged(&d_camera, sizeof(camera *)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2, d_texture_data, d_inxyn);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    hittable **shape;
    hittable **shape_list;
    int num_shape = 2;
    checkCudaErrors(hipMallocManaged(&shape, num_shape * sizeof(hittable *)));
    checkCudaErrors(hipMallocManaged(&shape_list, sizeof(hittable *))); 
    create_shape<<<1, 1>>>(shape, shape_list, num_shape);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // render buffer
    dim3 blocks( (nx - 1) / tx + 1, (ny - 1) / ty + 1);
    dim3 threads(tx, ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, shape_list, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    cerr << "took " << timer_seconds << " seconds to calculate.\n";

    // Output FB as Image
    cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; --j)
    {
        cerr << "\ralready writing: " << (int)(((double)(ny - j) / ny) * 100) << "%" << flush;
        for (int i = 0; i < nx; ++i)
        {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * clamp(fb[pixel_index].x(), 0.0, 0.999));
            int ig = int(255.99 * clamp(fb[pixel_index].y(), 0.0, 0.999));
            int ib = int(255.99 * clamp(fb[pixel_index].z(), 0.0, 0.999));
            cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list, d_world, d_camera, shape);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();

    cerr << "\nDone." << endl;
    return 0;
}