#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#define STB_IMAGE_IMPLEMENTATION        // ����STB_IMAGE_IMPLEMENTATION��ͷ�ļ�ֻ������������Դ�룬���ڽ�ͷ�ļ���Ϊ.cpp�ļ�
                                        // �������"�޷��������ⲿ���� stbi_load������ main �������˸÷���"
#include "./stb-master/stb_image.h"
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "aarect.h"
#include "box.h"
#include "moving_sphere.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"

using namespace std;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, const char *const func, const char *const file, const int line)
{
    if (result)
    {
        cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << "'" << func << "\n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ vec3 ray_color(const ray& r, const vec3& background, hittable **d_world, hiprandState *local_rand_state)
{
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    vec3 cur_emitted = vec3(0, 0, 0);
    for(int i = 0; i < 50; i++) 
    {
        hit_record rec;
        if ((*d_world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) 
        {
            ray scattered;
            vec3 attenuation;
            vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
            float pdf;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, pdf, local_rand_state)) 
            {
                cur_emitted = cur_emitted + cur_attenuation * emitted;
                cur_attenuation = cur_attenuation * attenuation * rec.mat_ptr->scattering_pdf(r, rec, scattered) / pdf;
                cur_ray = scattered;
                // ���Է����ʣ�����������Ȼ�ļ��ܴ�򲻿�
                /*printf("attenuation: %f, %f, %f\n", attenuation[0], attenuation[1], attenuation[2]);
                printf("current att: %f, %f, %f\n", cur_attenuation[0], cur_attenuation[1], cur_attenuation[2]);
                printf("scatter_pdf: %f\n", rec.mat_ptr->scattering_pdf(r, rec, scattered));
                printf("pdf        : %f\n", pdf);*/
            }
            else    
            {
                if (i == 0)
                    return emitted;
                return (cur_emitted + cur_attenuation * emitted); 
            }
        }
        else 
        {
            if (i == 0)
                return background;
            return cur_emitted + cur_attenuation * background;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) 
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(2020, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i > max_x || j > max_y) return;
    int pixel_index = j * max_x + i;
    // Each thread get same seed, a different sequence number, no offset
    hiprand_init(2020, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam,
                hittable **d_world, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = (j * max_x + i);
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    vec3 background(0, 0, 0);
    for (int s = 0; s < ns; ++s)
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += ray_color(r, background, d_world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);      // gamma corrected
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
    /*printf("block index : %d, %d\n", blockIdx.x, blockIdx.y);
    printf("thread index : %d, %d\n", threadIdx.x, threadIdx.y);*/
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera, 
            int nx, int ny, hiprandState *rand_state, unsigned char* d_texture_data, int* d_inxyn) 
{
    // d_world��һ��hittable_list���̳���hittable���洢�����еĿ���ײ���壬ͨ������d_world��hit����ʵ�ֶ���������ı������ҵ�������壬
    // �ṩ��һ����󣬷������Ҫ��ray_color���ֶ�չ�����б���
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {
        hiprandState local_rand_state = *rand_state;

        material* red = new lambertian(new const_texture(vec3(0.65, 0.05, 0.05)));
        material* white = new lambertian(new const_texture(vec3(0.73, 0.73, 0.73)));
        material* green = new lambertian(new const_texture(vec3(0.12, 0.45, 0.15)));

        material* light = new diffuse_light(new const_texture(vec3(15, 15, 15)));   // (15,15,15)

        material* ima = new lambertian(new image_texture(d_texture_data, d_inxyn[0], d_inxyn[1]));
        material* noise = new lambertian(new noise_texture(&local_rand_state, 0.2));

        d_list[0] = new flip_face(new yz_rect(green, 0, 555, 0, 555, 555));
        d_list[1] = new yz_rect(red, 0, 555, 0, 555, 0);
        d_list[2] = new flip_face(new xz_rect(light, 213, 343, 227, 332, 554));    // ��Դ (150, 400, 150, 400, 554)
        d_list[3] = new xz_rect(white, 0, 555, 0, 555, 0);
        d_list[4] = new flip_face(new xz_rect(white, 0, 555, 0, 555, 555));
        d_list[5] = new flip_face(new xy_rect(white, 0, 555, 0, 555, 555));

        // ����ת��ƽ�ƣ������޷��õ���ȷ��λ�ã�ԭ����ת����������y��������Ҫ������Ϊ��ת��������������غϣ�
        hittable* box1 = new box(white, vec3(0, 0, 0), vec3(165, 165, 165));    /// (0,0,0) (165,165,165)
        box1 = new rotate_y(box1, -18);
        d_list[6] = new translate(box1, vec3(130, 0, 65));  //(130,0,65)
        hittable* box2 = new box(white, vec3(0, 0, 0), vec3(165, 330, 165));      /// (0,0,0) (165,330,165)
        box2 = new rotate_y(box2, 15);
        d_list[7] = new translate(box2, vec3(265, 0, 295)); // (265,0,295)
        //d_list[3] = new moving_sphere(noise, vec3(300, 300, 300), vec3(300, 300, 300), 0, 1, 80);

        *rand_state = local_rand_state;
        *d_world  = new hittable_list(d_list, 8);

        vec3 lookfrom(278, 278, -800);  // 278, 278, -800
        vec3 lookat(278 , 278, 0);
        float dist_to_focus = 10; (lookfrom-lookat).length();
        float aperture = 0; //0.1
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vec3(0,1,0),
                                 40,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus,
                                 0.0,
                                 1.0);
    }
}

__global__ void free_world(hittable **d_list, hittable **d_world, camera **d_camera)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {
        /*for (int i = 0; i < 6; ++i)
        {
            //delete d_list[i]->mat_ptr;
            delete d_list[i];
        }*/
        delete *d_world;
        delete *d_camera;
    }
}

int main()
{
    const int nx = 1200;
    const int ny = 1200;
    const int ns = 200;     // ÿ��������������(�����)
    int tx = 16, ty = 16;

    cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    cerr << "in " << tx << "x" << ty << " threads.\n";

    const int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged(&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMallocManaged(&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMallocManaged(&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1, 1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // ֻ����host���������stbi_load����ȡͼƬ
    int inx, iny, inn;
    int *d_inxyn;
    // string����ת����const char*
    //string image_name = "IMG_20200910_000256.jpg"
    //"60847663_p0.jpg"
    unsigned char* texture_data = stbi_load("IMG_20200910_000256.jpg", &inx, &iny, &inn, 0);
    unsigned char* d_texture_data;
    // ����ͼ��
    checkCudaErrors(hipMallocManaged(&d_texture_data, inx * iny * inn * sizeof(unsigned char)));
    checkCudaErrors(hipMemcpy(d_texture_data, texture_data, inx * iny * inn * sizeof(unsigned char), hipMemcpyHostToDevice));
    // ����ͼ��ߴ缰ͨ����
    checkCudaErrors(hipMallocManaged(&d_inxyn, 3 * sizeof(int)));
    checkCudaErrors(hipMemcpy(d_inxyn, &inx, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_inxyn + 1, &iny, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_inxyn + 2, &inn, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hittables & camera
    hittable **d_list;      
    hittable **d_world;     // ʹ��ָ���ָ���ԭ���ǣ��������ָ���creat_world��ᴫ�ݸ��������¸���ʱ��������Ч�ռ�
    camera **d_camera;
    int num_hittables = 8;
    checkCudaErrors(hipMallocManaged(&d_list, num_hittables * sizeof(hittable *)));
    checkCudaErrors(hipMallocManaged(&d_world, sizeof(hittable *)));
    checkCudaErrors(hipMallocManaged(&d_camera, sizeof(camera *)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2, d_texture_data, d_inxyn);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // render buffer
    dim3 blocks( (nx - 1) / tx + 1, (ny - 1) / ty + 1);
    dim3 threads(tx, ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    cerr << "took " << timer_seconds << " seconds to calculate.\n";

    // Output FB as Image
    cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; --j)
    {
        cerr << "\ralready writing: " << (int)(((double)(ny - j) / ny) * 100) << "%" << flush;
        for (int i = 0; i < nx; ++i)
        {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].x());
            int ig = int(255.99 * fb[pixel_index].y());
            int ib = int(255.99 * fb[pixel_index].z());
            cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();

    cerr << "\nDone." << endl;
    return 0;
}